#include "hip/hip_runtime.h"
//
// Average extreme spread of five-shot group assuming impact coordinates follow standard normal distribution
//
#include <string>
#include <vector>
#include <numeric>
#include <stdexcept>
#include <typeinfo>
#include <hip/hip_runtime.h>
#include <hip/hip_cooperative_groups.h>

namespace cg = cooperative_groups;
#include <hiprand.h>

using std::string;
using std::vector;

// First level of reduction
__device__ double reduce_sum(double in, cg::thread_block cta)
{
  extern __shared__ double sdata[];

  // Write to shared memory
  unsigned ltid = threadIdx.x;
  sdata[ltid] = in;
  cg::sync(cta);

  // Do reduction in shared memory
  for (unsigned s = blockDim.x / 2 ; s > 0 ; s >>= 1) {
    if (ltid < s) {
      sdata[ltid] += sdata[ltid + s];
    }
    cg::sync(cta);
  }

  return sdata[0];
}

// Estimator kernel
__global__ void computeValue(double* const results,
                             const double* const points,
                             const unsigned int numSims)
{
  // Handle to thread block group
  cg::thread_block cta = cg::this_thread_block();

  // Determine thread ID
  unsigned bid = blockIdx.x;
  unsigned tid = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned step = gridDim.x * blockDim.x;

  // Shift the input/output pointers
  const double* pointx = points + tid;
  const double* pointy = pointx + numSims;

  // Count the number of points which lie inside the unit quarter-circle
  double pointsInside = 0;

  for (unsigned int i = tid ; i < numSims ; i += step, pointx += step, pointy += step) {
    double x = *pointx;
    double y = *pointy;
    double l2norm2 = x * x + y * y;

    if (l2norm2 < 1) {
      pointsInside += 1;
    }
  }

  // Reduce within the block
  pointsInside = reduce_sum(pointsInside, cta);

  // Store the result
  if (threadIdx.x == 0) {
    results[bid] = pointsInside;
  }
}

double es_cuda(unsigned int numSims, unsigned int threadBlockSize, unsigned int seed)
{
  // Get device properties
  struct hipDeviceProp_t  deviceProperties;
  hipError_t cudaResult = hipGetDeviceProperties(&deviceProperties, 0);
  if (cudaResult != hipSuccess) {
    string msg("Could not get device properties: ");
    msg += hipGetErrorString(cudaResult);
    throw std::runtime_error(msg);
  }

  // Check precision is valid
  if (deviceProperties.major < 1 || (deviceProperties.major == 1 && deviceProperties.minor < 3)) {
    throw std::runtime_error("Device does not have double precision support");
  }

  // Attach to GPU
  cudaResult = hipSetDevice(0);
  if (cudaResult != hipSuccess) {
    string msg("Could not set CUDA device: ");
    msg += hipGetErrorString(cudaResult);
    throw std::runtime_error(msg);
  }

  // Determine how to divide the work between cores
  dim3 block;
  dim3 grid;
  block.x = threadBlockSize;
  grid.x  = (numSims + threadBlockSize - 1) / threadBlockSize;

  // Aim to launch around ten or more times as many blocks as there
  // are multiprocessors on the target device.
  unsigned blocksPerSM = 10;
  unsigned numSMs      = deviceProperties.multiProcessorCount;
  while (grid.x > 2 * blocksPerSM * numSMs) {
    grid.x >>= 1;
  }

  // Get computeValue function properties and check the maximum block size
  struct hipFuncAttributes funcAttributes;
  cudaResult = hipFuncGetAttributes(&funcAttributes, reinterpret_cast<const void*>(computeValue));
  if (cudaResult != hipSuccess) {
    string msg("Could not get function attributes: ");
    msg += hipGetErrorString(cudaResult);
    throw std::runtime_error(msg);
  }

  if (block.x > (unsigned)funcAttributes.maxThreadsPerBlock) {
    throw std::runtime_error("Block X dimension is too large for computeValue kernel");
  }

  // Check the dimensions are valid
  if (block.x > (unsigned)deviceProperties.maxThreadsDim[0]) {
    throw std::runtime_error("Block X dimension is too large for device");
  }
  if (grid.x > (unsigned)deviceProperties.maxGridSize[0]) {
    throw std::runtime_error("Grid X dimension is too large for device");
  }

  // Allocate memory for points
  // Each simulation has two random numbers to give X and Y coordinate
  double *d_points = 0;
  cudaResult = hipMalloc((void **)&d_points, 2 * numSims * sizeof(double));
  if (cudaResult != hipSuccess) {
    string msg("Could not allocate memory on device for random numbers: ");
    msg += hipGetErrorString(cudaResult);
    throw std::runtime_error(msg);
  }

  // Allocate memory for result
  // Each thread block will produce one result
  double* d_results = 0;
  cudaResult = hipMalloc((void**)&d_results, grid.x * sizeof(double));
  if (cudaResult != hipSuccess) {
    string msg("Could not allocate memory on device for partial results: ");
    msg += hipGetErrorString(cudaResult);
    throw std::runtime_error(msg);
  }

  // Generate random points in unit square
  hiprandStatus_t curandResult;
  hiprandGenerator_t prng;
  curandResult = hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_DEFAULT);

  if (curandResult != HIPRAND_STATUS_SUCCESS) {
    string msg("Could not create pseudo-random number generator: ");
    msg += curandResult;
    throw std::runtime_error(msg);
  }

  curandResult = hiprandSetPseudoRandomGeneratorSeed(prng, seed);
  if (curandResult != HIPRAND_STATUS_SUCCESS) {
    string msg("Could not set seed for pseudo-random number generator: ");
    msg += curandResult;
    throw std::runtime_error(msg);
  }

  curandResult = hiprandGenerateUniformDouble(prng, (double*)d_points, 2 * numSims);
  if (curandResult != HIPRAND_STATUS_SUCCESS) {
    string msg("Could not generate pseudo-random numbers: ");
    msg += curandResult;
    throw std::runtime_error(msg);
  }

  curandResult = hiprandDestroyGenerator(prng);
  if (curandResult != HIPRAND_STATUS_SUCCESS) {
    string msg("Could not destroy pseudo-random number generator: ");
    msg += curandResult;
    throw std::runtime_error(msg);
  }

  // Count the points inside unit quarter-circle
  computeValue<<<grid, block, block.x * sizeof(double)>>>(d_results, d_points, numSims);

  // Copy partial results back
  vector<double> results(grid.x);
  cudaResult = hipMemcpy(&results[0], d_results, grid.x * sizeof(double), hipMemcpyDeviceToHost);

  if (cudaResult != hipSuccess) {
    string msg("Could not copy partial results to host: ");
    msg += hipGetErrorString(cudaResult);
    throw std::runtime_error(msg);
  }

  // Complete sum-reduction on host
  double value = std::accumulate(results.begin(), results.end(), 0);

  // Determine the proportion of points inside the quarter-circle,
  // i.e. the area of the unit quarter-circle
  value /= numSims;

  // Value is currently an estimate of the area of a unit quarter-circle, so we can
  // scale to a full circle by multiplying by four. Now since the area of a circle
  // is pi * r^2, and r is one, the value will be an estimate for the value of pi.
  value *= 4;

  // Cleanup
  if (d_points) {
    hipFree(d_points);
    d_points = 0;
  }

  if (d_results) {
    hipFree(d_results);
    d_results = 0;
  }

  return value;
}
