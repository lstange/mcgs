#include "hip/hip_runtime.h"
//
// Average extreme spread of five-shot group assuming impact coordinates follow standard normal distribution
//
// Building:
//   nvcc -std=c++11 es_cuda.cu -o es_cuda -lcurand
//
// Running:
//   for run in {1..10}; do ./es_cuda 15 | tee -a es_cuda.csv; done
//
#include <string>
#include <vector>
#include <numeric>
#include <stdexcept>
#include <typeinfo>
#include <hip/hip_cooperative_groups.h>
#include <iostream>
#include <iomanip>
#include <stdexcept>
#include <hip/hip_runtime.h>
#include <math.h>
#include <chrono>
#include <hiprand.h>

namespace cg = cooperative_groups;

using std::string;
using std::vector;

// First level of reduction
__device__ double reduce_sum(double in, cg::thread_block cta)
{
  extern __shared__ double sdata[];

  // Write to shared memory
  unsigned ltid = threadIdx.x;
  sdata[ltid] = in;
  cg::sync(cta);

  // Do reduction in shared memory
  for (unsigned s = blockDim.x / 2 ; s > 0 ; s >>= 1) {
    if (ltid < s) {
      sdata[ltid] += sdata[ltid + s];
    }
    cg::sync(cta);
  }

  return sdata[0];
}

// Estimator kernel
__global__ void computeValue(double* const results,
                             const double* const points,
                             const unsigned numGroups)
{
  // Handle to thread block group
  cg::thread_block cta = cg::this_thread_block();

  // Determine thread ID
  unsigned bid = blockIdx.x;
  unsigned tid = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned step = gridDim.x * blockDim.x;

  // Shift the input/output pointers
  const double* pointx = points + tid;
  const double* pointy = pointx + 5 * numGroups;

  // Sum up extreme spread of all groups
  double sum = 0;

  for (unsigned i = tid ; i < numGroups; i += step, pointx += step * 5, pointy += step * 5) {

    // Pairwise distances
    double dx[10], dy[10];

    // Unroll nested comparison loops
    dx[0] = pointx[0] - pointx[1]; dy[0] = pointy[0] - pointy[1];
    dx[1] = pointx[0] - pointx[2]; dy[1] = pointy[0] - pointy[2];
    dx[2] = pointx[0] - pointx[3]; dy[2] = pointy[0] - pointy[3];
    dx[3] = pointx[0] - pointx[4]; dy[3] = pointy[0] - pointy[4];
    dx[4] = pointx[1] - pointx[2]; dy[4] = pointy[1] - pointy[2];
    dx[5] = pointx[1] - pointx[3]; dy[5] = pointy[1] - pointy[3];
    dx[6] = pointx[1] - pointx[4]; dy[6] = pointy[1] - pointy[4];
    dx[7] = pointx[2] - pointx[3]; dy[7] = pointy[2] - pointy[3];
    dx[8] = pointx[2] - pointx[4]; dy[8] = pointy[2] - pointy[4];
    dx[9] = pointx[3] - pointx[4]; dy[9] = pointy[3] - pointy[4];

    double max_d2 = 0;
    for (unsigned j = 0; j < 10; j++) {
      auto candidate_d2 = dx[j] * dx[j] + dy[j] * dy[j];
      max_d2 = max(max_d2, candidate_d2);
    }
    double es = sqrt(max_d2);
    sum += es;
  }

  // Reduce within the block
  sum = reduce_sum(sum, cta);

  // Store the result
  if (threadIdx.x == 0) {
    results[bid] = sum;
  }
}

double es_cuda(unsigned power_of_4, unsigned seed)
{
  // Get device properties
  struct hipDeviceProp_t deviceProperties;
  hipError_t cudaResult = hipGetDeviceProperties(&deviceProperties, 0);
  if (cudaResult != hipSuccess) {
    string msg("Could not get device properties: ");
    msg += hipGetErrorString(cudaResult);
    throw std::runtime_error(msg);
  }

  // Check precision is valid
  if (deviceProperties.major < 1 || (deviceProperties.major == 1 && deviceProperties.minor < 3)) {
    throw std::runtime_error("Device does not have double precision support");
  }

  // Check requested size is valid
  const unsigned threadBlockSize = 128;
  if (threadBlockSize > (deviceProperties.maxThreadsPerBlock)) {
    throw std::runtime_error("Thread block size is greater than maxThreadsPerBlock");
  }
  dim3 block;
  block.x = threadBlockSize;

  // Attach to GPU
  cudaResult = hipSetDevice(0);
  if (cudaResult != hipSuccess) {
    string msg("Could not set CUDA device: ");
    msg += hipGetErrorString(cudaResult);
    throw std::runtime_error(msg);
  }

  // Aim to launch around ten or more times as many blocks as there
  // are multiprocessors on the target device.
  dim3 grid;
  const unsigned numGroups = 1 << (2 * power_of_4);
  grid.x  = numGroups / threadBlockSize;
  while (grid.x > 20 * deviceProperties.multiProcessorCount) {
    grid.x >>= 1;
  }

  // Get computeValue function properties and check the maximum block size
  struct hipFuncAttributes funcAttributes;
  cudaResult = hipFuncGetAttributes(&funcAttributes, reinterpret_cast<const void*>(computeValue));
  if (cudaResult != hipSuccess) {
    string msg("Could not get function attributes: ");
    msg += hipGetErrorString(cudaResult);
    throw std::runtime_error(msg);
  }

  if (block.x > (unsigned)funcAttributes.maxThreadsPerBlock) {
    throw std::runtime_error("Block X dimension is too large for computeValue kernel");
  }

  // Check the dimensions are valid
  if (block.x > (unsigned)deviceProperties.maxThreadsDim[0]) {
    throw std::runtime_error("Block X dimension is too large for device");
  }
  if (grid.x > (unsigned)deviceProperties.maxGridSize[0]) {
    throw std::runtime_error("Grid X dimension is too large for device");
  }

  // Allocate memory for points
  // Each simulation has ten random numbers to give five pairs of X and Y coordinates
  double* d_points = 0;
  cudaResult = hipMalloc((void **)&d_points, 10 * numGroups * sizeof(double));
  if (cudaResult != hipSuccess) {
    string msg("Could not allocate memory on device for random numbers: ");
    msg += hipGetErrorString(cudaResult);
    throw std::runtime_error(msg);
  }

  // Allocate memory for result
  // Each thread block will produce one result
  double* d_results = 0;
  cudaResult = hipMalloc((void**)&d_results, grid.x * sizeof(double));
  if (cudaResult != hipSuccess) {
    string msg("Could not allocate memory on device for partial results: ");
    msg += hipGetErrorString(cudaResult);
    throw std::runtime_error(msg);
  }

  // Generate random points
  hiprandStatus_t curandResult;
  hiprandGenerator_t prng;
  curandResult = hiprandCreateGenerator(&prng,  HIPRAND_RNG_PSEUDO_DEFAULT);
  if (curandResult != HIPRAND_STATUS_SUCCESS) {
    string msg("Could not create pseudo-random number generator: ");
    msg += curandResult;
    throw std::runtime_error(msg);
  }

  curandResult = hiprandSetPseudoRandomGeneratorSeed(prng, seed);
  if (curandResult != HIPRAND_STATUS_SUCCESS) {
    string msg("Could not set seed for pseudo-random number generator: ");
    msg += curandResult;
    throw std::runtime_error(msg);
  }

  curandResult = hiprandGenerateNormalDouble(prng, (double*)d_points, 10 * numGroups, 0, 1);
  if (curandResult != HIPRAND_STATUS_SUCCESS) {
    string msg("Could not generate pseudo-random numbers: ");
    msg += curandResult;
    throw std::runtime_error(msg);
  }

  curandResult = hiprandDestroyGenerator(prng);
  if (curandResult != HIPRAND_STATUS_SUCCESS) {
    string msg("Could not destroy pseudo-random number generator: ");
    msg += curandResult;
    throw std::runtime_error(msg);
  }

  // Calculate and average group size
  computeValue<<<grid, block, block.x * sizeof(double)>>>(d_results, d_points, numGroups);

  // Copy the results back to host
  vector<double> results(grid.x);
  cudaResult = hipMemcpy(&results[0], d_results, grid.x * sizeof(double), hipMemcpyDeviceToHost);
  if (cudaResult != hipSuccess) {
    string msg("Could not copy results to host: ");
    msg += hipGetErrorString(cudaResult);
    throw std::runtime_error(msg);
  }

  // Complete sum-reduction
  double sum = std::accumulate(results.begin(), results.end(), double(0));

  // Cleanup
  if (d_points) {
    hipFree(d_points);
  }
  if (d_results) {
    hipFree(d_results);
  }

  // Divide sum by count to get the average
  return sum / numGroups;
}

int main(int argc, char **argv)
{
  unsigned power_of_4 = 12;
  if (argc == 2) {
    power_of_4 = atoi(argv[1]);
  }
  unsigned nt = 12;
  if (power_of_4 > 12) {
    nt <<= 2 * (power_of_4 - 12);
    power_of_4 = 12;
  }
  try {
    auto start_time = std::chrono::system_clock::now();
    double avg = 0, min = 100, max = 0;
    __uint128_t mcg128_state = time(NULL) | 1; // can be seeded to any odd number
    for (unsigned j = 0; j < nt; j++) {
      double r = es_cuda(power_of_4, (unsigned)(mcg128_state >> 64));
      avg += r;
      min = fmin(r, min);
      max = fmax(r, max);
      mcg128_state *= 0xda942042e4dd58b5ULL;
    }
    avg /= nt;
    auto end_time = std::chrono::system_clock::now();
    std::chrono::duration<double> seconds = end_time - start_time;
    std::cout.precision(14);
    std::cout << "code,threads,power_of_4,min,avg,max,time\n";
    std::cout << "CUDA," << nt << "," << power_of_4 << "," << min << "," << avg << "," << max << "," << seconds.count() << "\n";
  } catch (std::runtime_error &e) { // es_cuda() can throw runtime exceptions
    fprintf(stderr, "runtime error (%s)\n", e.what());
    return(EXIT_FAILURE);
  }
  return(EXIT_SUCCESS);
}
