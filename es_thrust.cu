#include "hip/hip_runtime.h"
// sudo apt install nvidia-cuda-toolkit
// nvcc -O3 -std=c++11 es_thrust.cu -o es_thrust
//
#include <thrust/iterator/counting_iterator.h>
#include <thrust/functional.h>
#include <thrust/transform_reduce.h>
#include <iostream>
#include <iomanip>
#include <cmath>
#include <chrono>

const unsigned power_of_4 = 8;
const unsigned numGroups = 1 << 2 * power_of_4;

__constant__ uint64_t g_seed[1];

__device__
unsigned hash(unsigned a)
{
  a = (a+0x7ed55d16) + (a<<12);
  a = (a^0xc761c23c) ^ (a>>19);
  a = (a+0x165667b1) + (a<<5);
  a = (a+0xd3a2646c) ^ (a<<9);
  a = (a+0xfd7046c5) + (a<<3);
  a = (a^0xb55a4f09) ^ (a>>16);
  return a;
}

// Adopted from http://prng.di.unimi.it/xoshiro256plus.c
__device__
double xoshiro256plus(uint64_t s[4]) {
  const uint64_t result = s[0] + s[3];
  const uint64_t t = s[1] << 17;
  s[2] ^= s[0];
  s[3] ^= s[1];
  s[1] ^= s[2];
  s[0] ^= s[3];
  s[2] ^= t;
  s[3] = (s[3] << 45) | (s[3] >> (64 - 45));
  // random_real53() from http://mumble.net/~campbell/tmp/random_real.c
  return (double)(result >> 11) / (1ULL << 53);
}

struct estimate_es : public thrust::unary_function<unsigned, double>
{
  __device__
  double operator()(unsigned thread_id)
  {
    __uint64_t prng_state[4];
    __uint64_t mask = hash(thread_id);
    for (unsigned i = 0; i < 4; i++) {
      mask = (mask << 32) | hash(mask);
      mask = (mask << 32) | hash(mask);
      prng_state[i] = g_seed[0] ^ mask;
    }
    double sum = 0;
    for (unsigned group = 0; group < numGroups; group++) {
      double x[5]; double y[5];
      for (unsigned shot = 0; shot < 5; shot++) {
        auto u = xoshiro256plus(prng_state);
        auto v = xoshiro256plus(prng_state);
        
        // Box-Muller transform
        auto r = sqrt(-2 * log(u));
        auto theta = 2 * M_PI * v;
        x[shot] = r * cos(theta);
        y[shot] = r * sin(theta);
      }
      double es2 = 0;
      for (unsigned i = 0; i < 4; i++) {
        for (unsigned j = i + 1; j < 5; j++) {
          double dx = x[i] - x[j];
          double dy = y[i] - y[j];
          double d2 = dx * dx + dy * dy;
          if (es2 < d2) {
            es2 = d2;
          }
        }
      }
      sum += sqrt(es2);
    }
    return sum / numGroups;
  }
};

int main(void)
{
  auto start_time = std::chrono::system_clock::now();
  __uint128_t mcg128_state = ((__uint128_t)time(NULL) << 1) | 1;
  double avg = 0, min = 100, max = 0;
  const unsigned nt = 12;
  for (unsigned j = 0; j < nt; j++) {
    // MCG 128 PRNG from http://www.pcg-random.org/posts/on-vignas-pcg-critique.html
    mcg128_state *= 0xda942042e4dd58b5ULL;
    uint64_t seed[1] = {(uint64_t)(mcg128_state >> 64)};
    hipMemcpyToSymbol(HIP_SYMBOL(g_seed), seed, sizeof(uint64_t));
    double r = thrust::transform_reduce(thrust::counting_iterator<unsigned>(0),
                                        thrust::counting_iterator<unsigned>(numGroups),
                                        estimate_es(),
                                        double(0),
                                        thrust::plus<double>());
    avg += r / numGroups;
    min = fmin(r / numGroups, min);
    max = fmax(r / numGroups, max);
  }
  avg /= nt;
  auto end_time = std::chrono::system_clock::now();
  std::chrono::duration<double> seconds = end_time - start_time;
  std::cout.precision(14);
  std::cout << "code,threads,power_of_4,min,avg,max,time\n";
  std::cout << "CUDA thrust," << nt << "," << power_of_4 << "," << min << "," << avg << "," << max << "," << seconds.count() << "\n";
  return 0;
}
